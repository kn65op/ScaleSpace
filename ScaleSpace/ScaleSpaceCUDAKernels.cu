#include "hip/hip_runtime.h"
#include "ScaleSpaceCUDAKernels.h"

#include <opencv2\core\core.hpp>

__global__ void setLowToZero(cv::gpu::DevMem2Df in, cv::gpu::DevMem2Df out)
{
  int row = threadIdx.y;
  int col = threadIdx.x;
  float input = ((float*)in.ptr(row))[col];
  if (in.ptr(row)[col] < 1e-5)
  {
    ((float*)out.ptr(row))[col] = -3e4f;
  }
  else
  {
    ((float*)out.ptr(row))[col] = 4e4f;
    //out.ptr(row)[col] = in.ptr(row)[col];
  }
}

void __host__ setMatToZero(cv::gpu::GpuMat & in, cv::gpu::GpuMat & out)
{
  setLowToZero<<<in.cols, in.rows>>>(in, out);
}