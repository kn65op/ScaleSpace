#include "hip/hip_runtime.h"
#include "ScaleSpaceCUDAKernels.h"

#include <opencv2\core\core.hpp>

__global__ void setLowToZero(cv::gpu::PtrStepSzf in, cv::gpu::PtrStepSzf out)
{
  int row = blockIdx.y;
  int col = blockIdx.x;
  float input = ((float*)in.ptr(row))[col];
  if (in.ptr(row)[col] < 1e-5)
  {
    ((float*)out.ptr(row))[col] = -1e4f;
  }
  else
  {
    ((float*)out.ptr(row))[col] = 1e4f;
    //out.ptr(row)[col] = in.ptr(row)[col];
  }
}

void __host__ setMatToZero(cv::gpu::GpuMat & in, cv::gpu::GpuMat & out)
{
  dim3 co(1, 1);  
  dim3 cos(in.cols, in.rows);
  setLowToZero<<<cos, co>>>(in, out);
}